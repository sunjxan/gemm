#include "hip/hip_runtime.h"
#include "common.hpp"

// 双缓冲/预取，原理是指令级并行，如果指令之间相互独立，没有读写依赖，
// 线程读写操作的顺序和指令在代码中出现的顺序不一定相同
// __syncthreads函数可以确保该线程块在障碍点之前的读写操作已经完成
// __threadfence_block函数不进行障碍同步，只是挂起当前线程，直到之前的写入操作刷新完毕
// __threadfence_block/__threadfence/__threadfence_system对应刷新的内存和缓存层次不断提升

// block_shape应能整除M、K、N，unit应能整除K
// unit取简单情况等于block_shape，一次取满
constexpr size_t block_shape = 32, unit = block_shape;

__global__ void kernel(const real (*A)[K], const real (*B)[N], real (*C)[N])
{
    unsigned ty = threadIdx.y, iy = blockIdx.y * block_shape + ty;
    unsigned tx = threadIdx.x, ix = blockIdx.x * block_shape + tx;

    __shared__ real s_a[2][block_shape][unit], s_b[2][unit][block_shape];

    real sum = 0.0;

    // 取第一部分
    unsigned smem_stage_idx = 0;
    // 安培之前的架构，从全局内存转移到共享内存会经过寄存器中转
    s_a[smem_stage_idx][ty][tx] = A[iy][tx];
    s_b[smem_stage_idx][ty][tx] = B[ty][ix];
    // 协同拷贝，等待拷贝结束
    __syncthreads();

    // 调整循环下标
    for (size_t i = 1; i <= K / unit; ++i) {
        if (i != K / unit) {
            // 在A中拷贝的列序col_a，在B中拷贝的行序row_b
            size_t col_a = i * unit + tx, row_b = i * unit + ty;
            // 覆盖上一轮迭代计算使用的共享内存
            s_a[smem_stage_idx ^ 1][ty][tx] = A[iy][col_a];
            s_b[smem_stage_idx ^ 1][ty][tx] = B[row_b][ix];
        }
        for (size_t j = 0; j < unit; ++j) {
            sum += s_a[smem_stage_idx][ty][j] * s_b[smem_stage_idx][j][tx];
        }
        if (i != K / unit) {
            // 避免在共享内存使用之前被修改
            __syncthreads();
        }
        // 切换目标缓冲区
        smem_stage_idx ^= 1;
    }
    C[iy][ix] = sum;
}

void matmul(const real *A, const real *B, real *C)
{
    const real (*nA)[K] = reinterpret_cast<decltype(nA)>(A);
    const real (*nB)[N] = reinterpret_cast<decltype(nB)>(B);
    real (*nC)[N] = reinterpret_cast<decltype(nC)>(C);

    // 线程块是正方形
    dim3 block_size(block_shape, block_shape);
    // N是列对应x，M是行对应y
    dim3 grid_size(N / block_shape, M / block_shape);
    kernel<<<grid_size, block_size>>>(nA, nB, nC);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());
}

int main()
{
    launch_gpu();
    return 0;
}