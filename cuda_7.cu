#include "hip/hip_runtime.h"
#include "common.hpp"

// 让最后一个小迭代为下一轮预取数据

// block_shape应能整除M、K、N，block_unit应能整除K
constexpr size_t block_shape = 128, block_unit = 8;
// thread_shape应能整除block_shape和block_unit
constexpr size_t thread_shape = 8, block_dim = block_shape / thread_shape;
// thread_unit应能整除block_unit
constexpr size_t thread_unit = 1;

__global__ void kernel(const real (*A)[K], const real (*B)[N], real (*C)[N])
{

    unsigned ty = threadIdx.y, iy = blockIdx.y * block_shape + ty;
    unsigned tx = threadIdx.x, ix = blockIdx.x * block_shape + tx;

    __shared__ real s_a[2][block_shape][block_unit], s_b[2][block_unit][block_shape];

    // 不是协同拷贝，每个线程拷贝自己所需的数据，不同线程会重复拷贝数据
    real r_a[2][thread_shape][thread_unit], r_b[2][thread_unit][thread_shape];

    real sum[thread_shape][thread_shape];
    for (size_t p = 0; p < thread_shape; ++p) {
        for (size_t q = 0; q < thread_shape; ++q) {
            sum[p][q] = 0.0;
        }
    }

    // 取第一部分
    unsigned smem_stage_idx = 0;
    for (size_t j = 0; j < thread_shape; ++j) {
        for (size_t k = 0; tx + k * block_dim < block_unit; ++k) {
            // 安培之前的架构，从全局内存转移到共享内存会经过寄存器中转
            s_a[smem_stage_idx][ty + j * block_dim][tx + k * block_dim] =
                A[iy + j * block_dim][tx + k * block_dim];
        }
    }
    for (size_t k = 0; ty + k * block_dim < block_unit; ++k) {
        for (size_t j = 0; j < thread_shape; ++j) {
            s_b[smem_stage_idx][ty + k * block_dim][tx + j * block_dim] =
                B[ty + k * block_dim][ix + j * block_dim];
        }
    }
    // 协同拷贝，等待拷贝结束
    __syncthreads();

    // 取第一部分
    unsigned reg_stage_idx = 0;
    for (size_t p = 0; p < thread_shape; ++p) {
        for (size_t q = 0; q < thread_unit; ++q) {
            r_a[reg_stage_idx][p][q] = s_a[smem_stage_idx][ty + p * block_dim][q];
        }
    }
    for (size_t q = 0; q < thread_unit; ++q) {
        for (size_t p = 0; p < thread_shape; ++p) {
            r_b[reg_stage_idx][q][p] = s_b[smem_stage_idx][q][tx + p * block_dim];
        }
    }

    // 调整循环下标
    for (size_t i = 1; i <= K / block_unit; ++i) {
        // 调整循环下标
        // 展开复杂的内层循环
        #pragma unroll
        for (size_t j = 1; j <= block_unit / thread_unit; ++j) {
            // 提前到最后一次小迭代之前，切换到下一批次共享内存
            if (j == block_unit / thread_unit) {
                if (i != K / block_unit) {
                    // 避免在共享内存使用之前被修改
                    __syncthreads();
                }
                // 切换目标缓冲区
                smem_stage_idx ^= 1;
            }
            if (!(i == K / block_unit && j == block_unit / thread_unit)) {
                // 最后一次小迭代取下一批次共享内存里的开头部分
                size_t nj = j != block_unit / thread_unit ? j : 0;
                for (size_t p = 0; p < thread_shape; ++p) {
                    for (size_t q = 0; q < thread_unit; ++q) {
                        r_a[reg_stage_idx ^ 1][p][q] =
                            s_a[smem_stage_idx][ty + p * block_dim][q + nj * thread_unit];
                    }
                }
                for (size_t q = 0; q < thread_unit; ++q) {
                    for (size_t p = 0; p < thread_shape; ++p) {
                        r_b[reg_stage_idx ^ 1][q][p] =
                            s_b[smem_stage_idx][q + nj * thread_unit][tx + p * block_dim];
                    }
                }
            }
            // 推迟到第一次小迭代的预取之后
            if (j == 1) {
                if (i != K / block_unit) {
                    // 在A中拷贝的列序col_a，在B中拷贝的行序row_b
                    size_t col_a = i * block_unit + tx, row_b = i * block_unit + ty;
                    // 覆盖上一轮迭代计算使用的共享内存
                    for (size_t p = 0; p < thread_shape; ++p) {
                        for (size_t q = 0; tx + q * block_dim < block_unit; ++q) {
                            s_a[smem_stage_idx ^ 1][ty + p * block_dim][tx + q * block_dim] =
                                A[iy + p * block_dim][col_a + q * block_dim];
                        }
                    }
                    for (size_t q = 0; ty + q * block_dim < block_unit; ++q) {
                        for (size_t p = 0; p < thread_shape; ++p) {
                            s_b[smem_stage_idx ^ 1][ty + q * block_dim][tx + p * block_dim]
                                = B[row_b + q * block_dim][ix + p * block_dim];
                        }
                    }
                }
            }
            for (size_t k = 0; k < thread_unit; ++k) {
                for (size_t p = 0; p < thread_shape; ++p) {
                    for (size_t q = 0; q < thread_shape; ++q) {
                        sum[p][q] += r_a[reg_stage_idx][p][k] * r_b[reg_stage_idx][k][q];
                    }
                }
            }
            // 切换目标缓冲区
            reg_stage_idx ^= 1;
        }
    }
    for (size_t p = 0; p < thread_shape; ++p) {
        for (size_t q = 0; q < thread_shape; ++q) {
            // 注意sum计算和传值的对应方式
            C[iy + p * block_dim][ix + q * block_dim] = sum[p][q];
        }
    }
}

void matmul(const real *A, const real *B, real *C)
{
    const real (*nA)[K] = reinterpret_cast<decltype(nA)>(A);
    const real (*nB)[N] = reinterpret_cast<decltype(nB)>(B);
    real (*nC)[N] = reinterpret_cast<decltype(nC)>(C);

    // 线程块是正方形
    dim3 block_size(block_dim, block_dim);
    // N是列对应x，M是行对应y
    dim3 grid_size(N / block_shape, M / block_shape);
    kernel<<<grid_size, block_size>>>(nA, nB, nC);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());
}

int main()
{
    launch_gpu();
    return 0;
}