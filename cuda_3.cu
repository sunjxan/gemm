#include "hip/hip_runtime.h"
#include "common.hpp"

// 一个线程完成4X4个线程的工作

// block_shape应能整除M、K、N，unit应能整除K
constexpr size_t block_shape = 32, unit = 16;
// thread_shape应能整除block_shape和unit
constexpr size_t thread_shape = 4, block_dim = block_shape / thread_shape;
constexpr size_t frag_size = unit / block_dim;

__global__ void kernel(const real (*A)[K], const real (*B)[N], real (*C)[N])
{
    unsigned ty = threadIdx.y, iy = blockIdx.y * block_shape + ty;
    unsigned tx = threadIdx.x, ix = blockIdx.x * block_shape + tx;

    __shared__ real s_a[block_shape][unit], s_b[unit][block_shape];

    real sum[thread_shape][thread_shape];
    for (size_t p = 0; p < thread_shape; ++p) {
        for (size_t q = 0; q < thread_shape; ++q) {
            sum[p][q] = 0.0;
        }
    }
    for (size_t i = 0; i < K / unit; ++i) {
        // 在A中拷贝的列序col_a，在B中拷贝的行序row_b
        size_t col_a = i * unit + tx, row_b = i * unit + ty;
        for (size_t j = 0; j < thread_shape; ++j) {
            for (size_t k = 0; k < frag_size; ++k) {
                // 安培之前的架构，从全局内存转移到共享内存会经过寄存器中转
                s_a[ty + j * block_dim][tx + k * block_dim] =
                    A[iy + j * block_dim][col_a + k * block_dim];
            }
        }
        for (size_t k = 0; k < frag_size; ++k) {
            for (size_t j = 0; j < thread_shape; ++j) {
                s_b[ty + k * block_dim][tx + j * block_dim] =
                    B[row_b + k * block_dim][ix + j * block_dim];
            }
        }
        // 协同拷贝，等待拷贝结束
        __syncthreads();
        for (size_t j = 0; j < unit; ++j) {
            for (size_t p = 0; p < thread_shape; ++p) {
                for (size_t q = 0; q < thread_shape; ++q) {
                    sum[p][q] += s_a[ty + p * block_dim][j] * s_b[j][tx + q * block_dim];
                }
            }
        }
        if (i != K / unit - 1) {
            // 避免在共享内存使用之前被修改
            __syncthreads();
        }
    }
    for (size_t p = 0; p < thread_shape; ++p) {
        for (size_t q = 0; q < thread_shape; ++q) {
            // 注意sum计算和传值的对应方式
            C[iy + p * block_dim][ix + q * block_dim] = sum[p][q];
        }
    }
}

void matmul(const real *A, const real *B, real *C)
{
    const real (*nA)[K] = reinterpret_cast<decltype(nA)>(A);
    const real (*nB)[N] = reinterpret_cast<decltype(nB)>(B);
    real (*nC)[N] = reinterpret_cast<decltype(nC)>(C);

    // 线程块是正方形
    dim3 block_size(block_dim, block_dim);
    // N是列对应x，M是行对应y
    dim3 grid_size(N / block_shape, M / block_shape);
    kernel<<<grid_size, block_size>>>(nA, nB, nC);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());
}

int main()
{
    launch_gpu();
    return 0;
}
