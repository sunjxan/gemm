#include "hip/hip_runtime.h"
#include <cstdio>

#include "common.hpp"

// 一个线程完成4X4个线程的工作

// block_shape应能整除M、K、N，block_unit应能整除K
constexpr size_t block_shape = 32, block_unit = 16;
// thread_shape应能整除block_shape和block_unit
constexpr size_t thread_shape = 4, block_dim = block_shape / thread_shape;
// thread_unit应能整除block_unit
constexpr size_t thread_unit = 2, frag_size = block_unit / block_dim;

__device__ void kernel_thread(const real (*A)[block_unit], const real (*B)[block_shape], real (*C)[thread_shape])
{
    unsigned ty = threadIdx.y, tx = threadIdx.x;

    // 不是协同拷贝，每个线程拷贝自己所需的数据，不同线程会重复拷贝数据
    real r_a[thread_shape][thread_unit], r_b[thread_unit][thread_shape];

    for (size_t i = 0; i < block_unit / thread_unit; ++i) {
        for (size_t j = 0; j < thread_shape; ++j) {
            for (size_t k = 0; k < thread_unit; ++k) {
                r_a[j][k] = A[ty + j * block_dim][i * thread_unit + k];
            }
        }
        for (size_t k = 0; k < thread_unit; ++k) {
            for (size_t j = 0; j < thread_shape; ++j) {
                r_b[k][j] = B[i * thread_unit + k][tx + j * block_dim];
            }
        }
        for (size_t j = 0; j < thread_unit; ++j) {
            for (size_t p = 0; p < thread_shape; ++p) {
                for (size_t q = 0; q < thread_shape; ++q) {
                    C[p][q] += r_a[p][j] * r_b[j][q];
                }
            }
        }
    }
}

__global__ void kernel(const real (*A)[K], const real (*B)[N], real (*C)[N])
{

    unsigned ty = threadIdx.y, iy = blockIdx.y * block_shape + ty;
    unsigned tx = threadIdx.x, ix = blockIdx.x * block_shape + tx;

    __shared__ real s_a[block_shape][block_unit], s_b[block_unit][block_shape];

    real sum[thread_shape][thread_shape], frag_a[thread_shape][frag_size], frag_b[frag_size][thread_shape];
    for (size_t p = 0; p < thread_shape; ++p) {
        for (size_t q = 0; q < thread_shape; ++q) {
            sum[p][q] = 0.0f;
        }
    }
    // 安培之前的架构，从全局内存转移到共享内存需要经过寄存器，并做块同步
    for (size_t i = 0; i < K / block_unit; ++i) {
        // 在A中拷贝的列序col_a，在B中拷贝的行序row_b
        size_t i_block_unit = i * block_unit, col_a = i_block_unit + tx, row_b = i_block_unit + ty;
        for (size_t j = 0; j < thread_shape; ++j) {
            for (size_t k = 0; k < frag_size; ++k) {
                frag_a[j][k] = A[iy + j * block_dim][col_a + k * block_dim];
            }
        }
        for (size_t k = 0; k < frag_size; ++k) {
            for (size_t j = 0; j < thread_shape; ++j) {
                frag_b[k][j] = B[row_b + k * block_dim][ix + j * block_dim];
            }
        }
        __syncthreads();
        for (size_t j = 0; j < thread_shape; ++j) {
            for (size_t k = 0; k < frag_size; ++k) {
                s_a[ty + j * block_dim][tx + k * block_dim] = frag_a[j][k];
            }
        }
        for (size_t k = 0; k < frag_size; ++k) {
            for (size_t j = 0; j < thread_shape; ++j) {
                s_b[ty + k * block_dim][tx + j * block_dim] = frag_b[k][j];
            }
        }
        __syncthreads();
        kernel_thread(s_a, s_b, sum);
    }
    for (size_t p = 0; p < thread_shape; ++p) {
        for (size_t q = 0; q < thread_shape; ++q) {
            // 注意sum计算和传值的对应方式
            C[iy + p * block_dim][ix + q * block_dim] = sum[p][q];
        }
    }
}

void gemm(const real *A, const real *B, real *C)
{
    const real (*nA)[K] = reinterpret_cast<decltype(nA)>(A);
    const real (*nB)[N] = reinterpret_cast<decltype(nB)>(B);
    real (*nC)[N] = reinterpret_cast<decltype(nC)>(C);

    // 线程块是正方形
    dim3 block_size(block_dim, block_dim);
    // N是列对应x，M是行对应y
    dim3 grid_size(N / block_shape, M / block_shape);
    kernel<<<grid_size, block_size>>>(nA, nB, nC);
    CHECK(hipGetLastError());
}

int main()
{
    launch_gpu();
    return 0;
}
