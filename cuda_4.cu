#include "hip/hip_runtime.h"
#include "common.hpp"

// 1. 运用分治思想，矩阵分块加载到共享内存后，子矩阵相乘可以继续提高存储层次进行分块
// 2. 使用寄存器缓存线程负责的子矩阵的计算数据，为达到最高使用效率，子矩阵应是正方形
// 3. 寄存器数量有限，从共享内存加载到寄存器计算的过程，应在K轴上分段进行

// block_shape应能整除M、K、N，block_unit应能整除K
constexpr size_t block_shape = 32, block_unit = 16;
// thread_shape应能整除block_shape和block_unit
constexpr size_t thread_shape = 4, block_dim = block_shape / thread_shape;
// thread_unit应能整除block_unit
constexpr size_t thread_unit = 1, frag_size = block_unit / block_dim;

__device__ void kernel_thread(const real (*A)[block_unit], const real (*B)[block_shape], real (*C)[thread_shape])
{
    unsigned ty = threadIdx.y, tx = threadIdx.x;

    // 不是协同拷贝，每个线程拷贝自己所需的数据，不同线程会重复拷贝数据
    real r_a[thread_shape][thread_unit], r_b[thread_unit][thread_shape];

    for (size_t i = 0; i < block_unit / thread_unit; ++i) {
        for (size_t j = 0; j < thread_shape; ++j) {
            for (size_t k = 0; k < thread_unit; ++k) {
                r_a[j][k] = A[ty + j * block_dim][k + i * thread_unit];
            }
        }
        for (size_t k = 0; k < thread_unit; ++k) {
            for (size_t j = 0; j < thread_shape; ++j) {
                r_b[k][j] = B[k + i * thread_unit][tx + j * block_dim];
            }
        }
        for (size_t j = 0; j < thread_unit; ++j) {
            for (size_t p = 0; p < thread_shape; ++p) {
                for (size_t q = 0; q < thread_shape; ++q) {
                    C[p][q] += r_a[p][j] * r_b[j][q];
                }
            }
        }
    }
}

__global__ void kernel(const real (*A)[K], const real (*B)[N], real (*C)[N])
{

    unsigned ty = threadIdx.y, iy = blockIdx.y * block_shape + ty;
    unsigned tx = threadIdx.x, ix = blockIdx.x * block_shape + tx;

    __shared__ real s_a[block_shape][block_unit], s_b[block_unit][block_shape];

    real sum[thread_shape][thread_shape];
    for (size_t p = 0; p < thread_shape; ++p) {
        for (size_t q = 0; q < thread_shape; ++q) {
            sum[p][q] = 0.0;
        }
    }
    for (size_t i = 0; i < K / block_unit; ++i) {
        // 在A中拷贝的列序col_a，在B中拷贝的行序row_b
        size_t col_a = i * block_unit + tx, row_b = i * block_unit + ty;
        for (size_t j = 0; j < thread_shape; ++j) {
            for (size_t k = 0; k < frag_size; ++k) {
                // 安培之前的架构，从全局内存转移到共享内存会经过寄存器中转
                s_a[ty + j * block_dim][tx + k * block_dim] =
                    A[iy + j * block_dim][col_a + k * block_dim];
            }
        }
        for (size_t k = 0; k < frag_size; ++k) {
            for (size_t j = 0; j < thread_shape; ++j) {
                s_b[ty + k * block_dim][tx + j * block_dim] =
                    B[row_b + k * block_dim][ix + j * block_dim];
            }
        }
        // 协同拷贝，等待拷贝结束
        __syncthreads();
        kernel_thread(s_a, s_b, sum);
        if (i != K / block_unit - 1) {
            // 避免在共享内存使用之前被修改
            __syncthreads();
        }
    }
    for (size_t p = 0; p < thread_shape; ++p) {
        for (size_t q = 0; q < thread_shape; ++q) {
            // 注意sum计算和传值的对应方式
            C[iy + p * block_dim][ix + q * block_dim] = sum[p][q];
        }
    }
}

void matmul(const real *A, const real *B, real *C)
{
    const real (*nA)[K] = reinterpret_cast<decltype(nA)>(A);
    const real (*nB)[N] = reinterpret_cast<decltype(nB)>(B);
    real (*nC)[N] = reinterpret_cast<decltype(nC)>(C);

    // 线程块是正方形
    dim3 block_size(block_dim, block_dim);
    // N是列对应x，M是行对应y
    dim3 grid_size(N / block_shape, M / block_shape);
    kernel<<<grid_size, block_size>>>(nA, nB, nC);
    CHECK(hipDeviceSynchronize());
}

int main()
{
    launch_gpu();
    return 0;
}
