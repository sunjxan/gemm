#include <cstdio>
#include <hipblas.h>

#include "common.cuh"

void gemm(const real *d_A, const real *d_B, real *d_C)
{
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));
    real alpha = 1.0f, beta = 0.0f;
    #ifdef USE_DP
        CHECK_CUBLAS(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K,
            &alpha, d_B, N, d_A, K, &beta, d_C, N));
    #else
        CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K,
            &alpha, d_B, N, d_A, K, &beta, d_C, N));
    #endif
    CHECK_CUBLAS(hipblasDestroy(handle));
}

int main()
{
    launch_gpu();
    return 0;
}
