#include <hipblas.h>

#include "common.hpp"

void matmul(const real *A, const real *B, real *C)
{
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));
    real alpha = 1.0, beta = 0.0;
    #ifdef USE_DP
        CHECK_CUBLAS(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K,
            &alpha, B, N, A, K, &beta, C, N));
    #else
        CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K,
            &alpha, B, N, A, K, &beta, C, N));
    #endif
    CHECK_CUBLAS(hipblasDestroy(handle));
}

int main()
{
    launch_gpu();
    return 0;
}
