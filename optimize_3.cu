#include "hip/hip_runtime.h"
#include "common.hpp"

// 使用向量读取指令LDS.128优化 Shared Memory 访问（对应 float4 数据类型），大幅减少访存指令的数量

#define FLOAT4(pointer) (reinterpret_cast<float4 *>(&(pointer))[0])
#define CFLOAT4(pointer) (reinterpret_cast<const float4 *>(&(pointer))[0])

// block_shape应能整除M、K、N，block_unit应能整除K，block_unit应是4的正整数倍
constexpr size_t block_shape = 128, block_unit = 8;
// thread_shape应能整除block_shape和block_unit
constexpr size_t thread_shape = 8, block_dim = block_shape / thread_shape;
// thread_unit应能整除block_unit
constexpr size_t thread_unit = 1;

__global__ void kernel(const real (*A)[K], const real (*B)[N], real (*C)[N])
{
    unsigned tid = threadIdx.x, ty = tid / block_dim, tx = tid % block_dim;
    unsigned by = blockIdx.y * block_shape, iy = by + ty;
    unsigned bx = blockIdx.x * block_shape, ix = bx + tx;

    __shared__ real s_a[2][block_shape][block_unit], s_b[2][block_unit][block_shape];

    // 不是协同拷贝，每个线程拷贝自己所需的数据，不同线程会重复拷贝数据
    real r_a[2][thread_shape][thread_unit], r_b[2][thread_unit][thread_shape];

    real sum[thread_shape][thread_shape];
    for (size_t p = 0; p < thread_shape; ++p) {
        for (size_t q = 0; q < thread_shape; ++q) {
            sum[p][q] = 0.0;
        }
    }

    // 取第一部分
    unsigned smem_stage_idx = 0;
    size_t bit = real_size == sizeof(float) ? 2 : 1;
    for (size_t i = tid << bit; i < block_shape * block_unit; i += blockDim.x << bit) {
        size_t j = i / block_unit, k = i % block_unit;
        // 安培之前的架构，从全局内存转移到共享内存会经过寄存器中转
        FLOAT4(s_a[smem_stage_idx][j][k]) = CFLOAT4(A[j + by][k]);

        k = i / block_shape, j = i % block_shape;
        FLOAT4(s_b[smem_stage_idx][k][j]) = CFLOAT4(B[k][j + bx]);
    }
    // 协同拷贝，等待拷贝结束
    __syncthreads();

    // 取第一部分
    unsigned reg_stage_idx = 0;
    for (size_t p = 0; p < thread_shape; ++p) {
        for (size_t q = 0; q < thread_unit; ++q) {
            r_a[reg_stage_idx][p][q] = s_a[smem_stage_idx][ty + p * block_dim][q];
        }
    }
    for (size_t q = 0; q < thread_unit; ++q) {
        for (size_t p = 0; p < thread_shape; ++p) {
            r_b[reg_stage_idx][q][p] = s_b[smem_stage_idx][q][tx + p * block_dim];
        }
    }

    // 调整循环下标
    for (size_t i = 1; i <= K / block_unit; ++i) {
        // 调整循环下标
        #pragma unroll
        for (size_t j = 1; j <= block_unit / thread_unit; ++j) {
            // 提前到最后一次小迭代之前，切换到下一批次共享内存
            if (j == block_unit / thread_unit) {
                if (i != K / block_unit) {
                    // 避免在共享内存使用之前被修改
                    __syncthreads();
                }
                // 切换目标缓冲区
                smem_stage_idx ^= 1;
            }
            if (!(i == K / block_unit && j == block_unit / thread_unit)) {
                // 最后一次小迭代取下一批次共享内存里的开头部分
                size_t nj = j != block_unit / thread_unit ? j : 0;
                for (size_t p = 0; p < thread_shape; ++p) {
                    for (size_t q = 0; q < thread_unit; ++q) {
                        r_a[reg_stage_idx ^ 1][p][q] =
                            s_a[smem_stage_idx][ty + p * block_dim][q + nj * thread_unit];
                    }
                }
                for (size_t q = 0; q < thread_unit; ++q) {
                    for (size_t p = 0; p < thread_shape; ++p) {
                        r_b[reg_stage_idx ^ 1][q][p] =
                            s_b[smem_stage_idx][q + nj * thread_unit][tx + p * block_dim];
                    }
                }
            }
            // 推迟到第一次小迭代的预取之后
            if (j == 1) {
                if (i != K / block_unit) {
                    // 覆盖上一轮迭代计算使用的共享内存
                    for (size_t r = tid << bit; r < block_shape * block_unit; r += blockDim.x << bit) {
                        size_t s = r / block_unit, t = r % block_unit;
                        // 安培之前的架构，从全局内存转移到共享内存会经过寄存器中转
                        FLOAT4(s_a[smem_stage_idx ^ 1][s][t]) = CFLOAT4(A[s + by][t + i * block_unit]);

                        t = r / block_shape, s = r % block_shape;
                        FLOAT4(s_b[smem_stage_idx ^ 1][t][s]) = CFLOAT4(B[t + i * block_unit][s + bx]);
                    }
                }
            }
            for (size_t k = 0; k < thread_unit; ++k) {
                for (size_t p = 0; p < thread_shape; ++p) {
                    for (size_t q = 0; q < thread_shape; ++q) {
                        sum[p][q] += r_a[reg_stage_idx][p][k] * r_b[reg_stage_idx][k][q];
                    }
                }
            }
            // 切换目标缓冲区
            reg_stage_idx ^= 1;
        }
    }
    for (size_t p = 0; p < thread_shape; ++p) {
        for (size_t q = 0; q < thread_shape; ++q) {
            // 注意sum计算和传值的对应方式
            C[iy + p * block_dim][ix + q * block_dim] = sum[p][q];
        }
    }
}

void matmul(const real *A, const real *B, real *C)
{
    const real (*nA)[K] = reinterpret_cast<decltype(nA)>(A);
    const real (*nB)[N] = reinterpret_cast<decltype(nB)>(B);
    real (*nC)[N] = reinterpret_cast<decltype(nC)>(C);

    // 线程块是正方形
    dim3 block_size(block_dim * block_dim);
    // N是列对应x，M是行对应y
    dim3 grid_size(N / block_shape, M / block_shape);
    kernel<<<grid_size, block_size>>>(nA, nB, nC);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());
}

int main()
{
    launch_gpu();
    return 0;
}